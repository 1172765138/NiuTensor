#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
 * Copyright (C) 2017, Natural Language Processing Lab, Northestern University. 
 * All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
* $Created by: XIAO Tong (email: xiaotong@mail.neu.edu.cn) 2018-04-25
*/

#include "HardTanH.h"
#include "HardTanH.cuh"
#include "Loss.cuh"
#include "CrossEntropy.cuh"
#include "../XDevice.h"

namespace nts{ // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/* 
hard tanh forward computation (Cuda kernel) 
y =  1    if x > 1
     x    if -1 <= x <= 1
    -1    if x < -1
>> x - input data array
>> y - output data array
>> size - size of input/output
*/
__global__ 
void KernelHardtanhCompute(DTYPE * x, DTYPE * y, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size){
        DTYPE p = x[i];
        if(p > (DTYPE)1.0)
            p = (DTYPE)1.0;
        else if(p < (DTYPE)-1.0)
            p = (DTYPE)-1.0;
        y[i] = p;
    }
}

/*
hard tanh forward computation (Cuda version) 
y =  1    if x > 1
     x    if -1 <= x <= 1
    -1    if x < -1
>> x - input tensor
>> y - output tensor
*/
void _CudaHardTanH(const XTensor * x, XTensor * y)
{
    if(x->dataType == DEFAULT_DTYPE && y->dataType == DEFAULT_DTYPE){

        CheckNTErrors(!x->isSparse && !y->isSparse, "The hard tanh activation function does not support sparse tensors.");
        CheckNTErrors(x->unitNum && y->unitNum, "The x vectors must be of the same length.");

        int gridSize[3], blockSize[3];

        GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

        int devIDBackup;
        ProtectCudaDev(x->devID, devIDBackup);

        KernelHardtanhCompute<<<dim3(gridSize[0]), dim3(blockSize[0])>>>((DTYPE*)x->data, (DTYPE*)y->data, x->unitNum);

        BacktoCudaDev(x->devID, devIDBackup);
    }
    else{
        ShowNTErrors("TODO!");
    }
}

/* 
hard tanh backward computation of dE/dx (Cuda kernel)

dy/dx = 1     if -1 <= x <= 1
        0     otherwise

>> dedy - dE/dy
>> dedx - dE/dx
>> gold - gold standard
>> y - y of the function
>> x - x of the function
>> size - size of y/x
*/
__global__ 
void KernelHardtanhBackward(DTYPE * dedy, DTYPE * dedx, DTYPE * gold, DTYPE * y, DTYPE * x, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size){
        DTYPE s = x[i];
        if(s > (DTYPE)1.0 || s < (DTYPE)-1.0)
            dedx[i] = 0;
        else
            dedx[i] = dedy[i];
    }
}

/*
backward computation (Cuda version)

dE/dx = dE/dy * dy/dx

hard tanh: y =  1    if x > 1
                x    if -1 <= x <= 1
               -1    if x< -1

   and dy/dx =  1    if -1 <= x <= 1
                0    otherwise

>> gold - gold standard to measure error (or loss)
>> y - output of the function
>> x - input of the function
>> dedy - dE/dy
>> dedx - dE/dx
>> lossName - type of loss function, e.g., cross entropy
*/
void _CudaHardTanHBackward(XTensor * gold, XTensor * y, XTensor * x, 
                           XTensor * dedy, XTensor * dedx,
                           LOSS_FUNCTION_NAME lossName)
{
    if(x->dataType == DEFAULT_DTYPE && y->dataType == DEFAULT_DTYPE){

        /* calculate dE/dy */
        if(lossName == CROSSENTROPY)
            _CudaCrossEntropyBackward(dedy, y, gold);
        else if(lossName != NOLOSS)
            _CudaLossBackward(dedy, gold, y, lossName);

        int gridSize[3], blockSize[3];

        GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

        int devIDBackup;
        ProtectCudaDev(x->devID, devIDBackup);

        /* dE/dx = dE/dy * dy/dx */
        KernelHardtanhBackward<<<dim3(gridSize[0]),dim3(blockSize[0])>>>
                               ((DTYPE*)dedy->data, 
                                (DTYPE*)dedx->data,
                                 gold == NULL ? NULL : (DTYPE*)gold->data, 
                                (DTYPE*)y->data, (DTYPE*)x->data, 
                                 x->unitNum);

        BacktoCudaDev(x->devID, devIDBackup);
    }
    else
        ShowNTErrors("TODO!");
}

#endif

} // namespace nts(NiuTrans.Tensor)