#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
 * Copyright (C) 2017, Natural Language Processing Lab, Northestern University. 
 * All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * $Created by: XIAO Tong (email: xiaotong@mail.neu.edu.cn) 2018-04-25
 */

#include "Sigmoid.h"
#include "Sigmoid.cuh"
#include "Loss.cuh"
#include "CrossEntropy.cuh"
#include "../XDevice.h"

#ifdef USE_CUDA

// the CUDA stuff
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#endif

namespace nts{ // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/* 
sigmoid function y = 1/(1+exp(-x))  (Cuda kernel) 
>> x - input data pointer
>> y - output data pointer
>> size - size of input/output
*/
__global__ 
void KernelSigmoidCompute(DTYPE * x, DTYPE * y, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size){
        y[i] = 1/(1+exp(-x[i]));
    }
}

/*
sigmoid function y = 1/(1+exp(-x)) (Cuda version)
>> x - input vector
>> y - result
*/
void _CudaSigmoid(const XTensor * x, XTensor * y)
{
    if(x->dataType == DEFAULT_DTYPE && y->dataType == DEFAULT_DTYPE){

        CheckNTErrors(!x->isSparse && !y->isSparse, "the activation function (rectify) does not support sparse matrices.");
        CheckNTErrors(x->unitNum && y->unitNum, "we require two vectors with the same length.");

        int gridSize[3], blockSize[3];

        GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

        int devIDBackup;
        ProtectCudaDev(x->devID, devIDBackup);

        KernelSigmoidCompute<<<dim3(gridSize[0]), dim3(blockSize[0])>>>((DTYPE*)x->data, (DTYPE*)y->data, x->unitNum);

        BacktoCudaDev(x->devID, devIDBackup);
    }
    else
        ShowNTErrors("TODO!");
}

/* 
sigmoid backward computation of dE/dx (Cuda kernel)

dE/ds = dE/dy * dy/dx

sigmoid: y = 1/(1+exp(-x))

   and dy/ds = y * (1 -y)

>> dedy - dE/dy
>> dedx - dE/ds
>> gold - gold standard
>> y - output of the function
>> x - input of the function
>> size - size of output/input
*/
__global__ 
void KernelSigmoidBackward(DTYPE * dedy, DTYPE * dedx, DTYPE * gold, DTYPE * y, DTYPE * x, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size){
        dedx[i] = dedy[i] * y[i] * ((DTYPE)1.0 - y[i]);
    }
}

/*
backward computation (Cuda version)

dE/ds = dE/dy * dy/dx

sigmoid: y = 1/(1+exp(-x))

   and dy/dx = y * (1 -y)

>> gold - gold standard to measure error (or loss)
>> y - output of the function
>> x - input of the function
>> dedy - dE/dy
>> dedx - dE/dx
>> lossName - type of loss function, e.g., cross entropy
*/
void _CudaSigmoidBackward(XTensor * gold, XTensor * y, XTensor * x, 
                          XTensor * dedy, XTensor * dedx,
                          LOSS_FUNCTION_NAME lossName)
{
    if(x->dataType == DEFAULT_DTYPE && y->dataType == DEFAULT_DTYPE){
        /* calculate dE/dy */
        if(lossName == CROSSENTROPY)
            _CudaCrossEntropyBackward(dedy, y, gold);
        else if(lossName != NOLOSS)
            _LossBackward(dedy, gold, y, lossName);

        
        int gridSize[3], blockSize[3];

        GDevs.GetCudaThread(x->devID, x->unitNum, gridSize, blockSize);

        int devIDBackup;
        ProtectCudaDev(x->devID, devIDBackup);

        /* dE/ds = dE/dy * dy/ds */
        KernelSigmoidBackward<<<dim3(gridSize[0]),dim3(blockSize[0])>>>
                              ((DTYPE*)dedy->data, 
                               (DTYPE*)dedx->data,
                                gold == NULL ? NULL : (DTYPE*)gold->data, 
                               (DTYPE*)y->data, (DTYPE*)x->data, 
                                x->unitNum);

        BacktoCudaDev(x->devID, devIDBackup);
    }
    else
        ShowNTErrors("TODO!");
}

#endif

} // namespace nts(NiuTrans.Tensor)