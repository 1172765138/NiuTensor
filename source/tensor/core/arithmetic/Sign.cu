#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
* Copyright (C) 2017, Natural Language Processing Lab, Northestern University.
* All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

/*
* $Created by: LI Yinqiao (li.yin.qiao.2012@hotmail.com) 2018-7-11
*/

#include "../../XDevice.h"
#include "../../XTensor.h"
#include "Sign.h"
#include "Sign.cuh"

namespace nts { // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA
/*
set each entry to its sign value (CUDA Kernel)
>> a - pointer to input data array
>> b - pointer to output data array
>> size - size of the data array
*/
__global__
void KernelSign(DTYPE * a, DTYPE * b, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size) {
        if (a[i] > 0)
            b[i] = 1.0F;
        else if (a[i] == 0)
            b[i] = 0.0F;
        else
            b[i] = -1.0F;
    }
}

/*
set each entry to its sign value with float16 data type value (CUDA Kernel)
This is for float16 computation
>> a - pointer to input data array
>> b - pointer to output data array
>> size - size of the data array
*/
__global__
void KernelSign(__half * a, __half * b, int size)
{
    return;
}

/*
set each entry to its sign value
>> a - input tensor we are processing
>> b - output tensor we are processing
*/
void _CudaSign(const XTensor * a, XTensor * b)
{
    CheckNTErrors((XTensor::IsSameShaped(a, b)), "Input tensors should have the same type!");
    CheckNTErrors((a->isSparse == false), "TODO!");

    int gridSize[3];
    int blockSize[3];

    GDevs.GetCudaThread(a->devID, a->unitNum, gridSize, blockSize);

    dim3 blocks(gridSize[0]);
    dim3 threads(blockSize[0]);

    int devIDBackup;
    ProtectCudaDev(a->devID, devIDBackup);

    if (a->dataType == DEFAULT_DTYPE) {
        KernelSign << <blocks, threads >> >((DTYPE*)a->data, (DTYPE*)b->data, a->unitNum);
    }
    else if (a->dataType == X_FLOAT16) {
        KernelSign << <blocks, threads >> >((__half*)a->data, (__half*)b->data, a->unitNum);
    }
    else {
        ShowNTErrors("TODO!");
    }

    BacktoCudaDev(a->devID, devIDBackup);
}

#endif // USE_CUDA
} // namespace nts(NiuTrans.Tensor)
