#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
* Copyright (C) 2017, Natural Language Processing Lab, Northestern University.
* All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

/*
* $Created by: XIAO Tong (email: xiaotong@mail.neu.edu.cn) 2018-04-24
*/

#include "../../XDevice.h"
#include "../../XTensor.h"
#include "SumByColumnTV.h"
#include "SumByColumnTV.cuh"

namespace nts { // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA

/*
summation of a tensor and a vector (column vector)
c_col = a_col  + b * \beta
>> a - a tensor
>> b - a vector with the same column size with a
>> c - where we put a+b. we save it in a
>> colNum - column number (of a block)
>> blockSize - size of a block
>> size - size of the entire data array
>> beta - the scaling factor
*/
__global__
void KernelADDByColumnTV(DTYPE * a, DTYPE * b, DTYPE * c, int colNum, int blockSize, int size, DTYPE beta)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= size)
        return;

    int offset = i % blockSize;
    int row = offset / colNum;

    c[i] = a[i] + b[row] * beta;
}

/*
summation of a tensor and a vector (column vector)
for each column a_col (in a block), we have
c_col = a_col + b * \beta
where b is a vector.

>> a - a tensor
>> b - a vector with the same column size with a
>> c - where we put a+b. we save it in a if c is NULL
>> beta - the scaling factor
*/
void _CudaSumByColumnTV(const XTensor * a, const XTensor * b, XTensor * c, DTYPE beta)
{
    CheckNTErrors((a && b && c), "Empty input tensors!");
    CheckNTErrors((XTensor::IsSameShaped(a, c)), "Unmatched tensors in addition!");
    CheckNTErrors((b->order == 2 && b->dimSizeRDI[0] == 1 && b->dimSizeRDI[1] == a->dimSizeRDI[1]),
                  "Illegal input vector size!");
    CheckNTErrors((a->dataType == DEFAULT_DTYPE && b->dataType == DEFAULT_DTYPE &&
                  c->dataType == DEFAULT_DTYPE), "TODO");

    int rowNum = a->dimSize[0];
    int colNum = a->dimSize[1];
    int blockNum = 1;
    for (int i = 2; i < a->order; i++)
        blockNum *= a->dimSizeRDI[i];

    int cudaGridSize[3];
    int cudaBlockSize[3];

    GDevs.GetCudaThread(c->devID, a->unitNum, cudaGridSize, cudaBlockSize);

    int devIDBackup;
    ProtectCudaDev(a->devID, devIDBackup);

    KernelADDByColumnTV << <dim3(cudaGridSize[0]), dim3(cudaBlockSize[0]) >> >
                          ((DTYPE*)a->data, (DTYPE*)b->data, (DTYPE*)c->data, colNum, rowNum * colNum, a->unitNum, beta);

    BacktoCudaDev(a->devID, devIDBackup);
}

#endif // USE_CUDA

} // namespace nts(NiuTrans.Tensor)